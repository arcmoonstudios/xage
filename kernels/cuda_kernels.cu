
#include <hip/hip_runtime.h>
// kernels/cuda_kernels.cu

extern "C" __global__ void vector_add(float* a, float* b, float* result, int num_elements) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_elements) {
        result[idx] = a[idx] + b[idx];
    }
}

extern "C" __global__ void matrix_multiply(float* a, float* b, float* c, int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < m && col < n) {
        float sum = 0.0f;
        for (int i = 0; i < k; ++i) {
            sum += a[row * k + i] * b[i * n + col];
        }
        c[row * n + col] = sum;
    }
}

extern "C" __global__ void relu_activation(float* data, int num_elements) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_elements) {
        data[idx] = fmaxf(0.0f, data[idx]);
    }
}

extern "C" __global__ void softmax(float* input, float* output, int num_elements) {
    __shared__ float max_val;
    __shared__ float sum;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float thread_max = -INFINITY;
    if (idx < num_elements) {
        thread_max = input[idx];
    }
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (threadIdx.x < offset) {
            thread_max = fmaxf(thread_max, __shfl_down_sync(0xffffffff, thread_max, offset));
        }
    }
    if (threadIdx.x == 0) {
        max_val = thread_max;
    }
    __syncthreads();
    float thread_sum = 0.0f;
    if (idx < num_elements) {
        output[idx] = expf(input[idx] - max_val);
        thread_sum = output[idx];
    }
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (threadIdx.x < offset) {
            thread_sum += __shfl_down_sync(0xffffffff, thread_sum, offset);
        }
    }
    if (threadIdx.x == 0) {
        sum = thread_sum;
    }
    __syncthreads();
    if (idx < num_elements) {
        output[idx] /= sum;
    }
}
